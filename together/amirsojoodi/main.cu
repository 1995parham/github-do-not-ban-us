
#include <hip/hip_runtime.h>
#define GRID_DIM 80000000 // Iran's Population
#define BLOCK_DIM 1024 // Most available threads of a GPU block

__global__ void protest_kernel(){
	printf("Thread %d: GitHub do not ban us!\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int main(){
	printf("GitHub is for everyone.\n");
	protest_kernel<<<GRID_DIM, BLOCK_DIM>>>();
	return 0;
}
