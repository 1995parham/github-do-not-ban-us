
#include <hip/hip_runtime.h>
#define GRID_DIM 80000000 // Iran's Population
#define BLOCK_DIM 1024 // Most available threads of a GPU block

__global__ void protest_kernel(){
	printf("Thread %d: Github do not ban us!\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int main(){
	printf("Github is for everyone.\n");
	protest_kernel<<<GRID_DIM, BLOCK_DIM>>>();
	return 0;
}
